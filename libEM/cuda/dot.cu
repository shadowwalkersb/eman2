#include "hip/hip_runtime.h"
#include "dot.h"

#include <stdio.h>
#include <vector>

using namespace std;

typedef vector<float> VF;

__global__
void kernel_hello() {
    printf("Hello! Idx: %d\n", threadIdx);
}

void cuda_hello() {
    kernel_hello<<<1,5>>>();
    hipDeviceSynchronize();
}

float cpp_dot(EMData &obj1, EMData &obj2) {
    VF v1(obj1.get_data_as_vector());
    VF v2(obj2.get_data_as_vector());

    float sum = 0.0f;
    for(int i=0; i<v1.size(); ++i)
        sum += v1[i] * v2[i];

    return sum;
}
