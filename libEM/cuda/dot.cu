#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void kernel_hello() {
    printf("Hello! Idx: %d\n", threadIdx);
}

void cuda_hello() {
    kernel_hello<<<1,5>>>();
    hipDeviceSynchronize();
}
