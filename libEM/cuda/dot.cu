#include "hip/hip_runtime.h"
#include "dot.h"

#include <stdio.h>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include "hipblas.h"

#define THREADSPERBLOCK 1024

using namespace std;

typedef vector<float> VF;

typedef thrust::host_vector<float> HV;
typedef thrust::device_vector<float> DV;

__global__
void kernel_hello() {
    printf("Hello! Idx: %d\n", threadIdx);
}

void cuda_hello() {
    kernel_hello<<<1,5>>>();
    hipDeviceSynchronize();
}

float cpp_dot(EMData &obj1, EMData &obj2) {
    VF v1(obj1.get_data_as_vector());
    VF v2(obj2.get_data_as_vector());

    float sum = 0.0f;
    for(int i=0; i<v1.size(); ++i)
        sum += v1[i] * v2[i];

    return sum;
}

__global__
void kernel_dot(float *v1, float *v2, int N, float *sum) {
    __shared__ float cc[THREADSPERBLOCK];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<N) {
        float mm = v1[i] * v2[i];
        cc[threadIdx.x] = v1[i] * v2[i];
//        printf(" %d %f %f %d %d %d\n", i, ss, *sum, blockDim.x, blockIdx.x, threadIdx.x);
//        atomicAdd(sum, mm);
//        *su += ss;
//        printf("sum= %f", *sum);
    }
    __syncthreads();

    for(int stride = blockDim.x/2; threadIdx.x < stride && stride>0; stride /=2) {
        cc[threadIdx.x] += cc[threadIdx.x + stride];
        __syncthreads();
    }
    
    if(threadIdx.x == 0)
        sum[blockIdx.x] = cc[0];
}

float cuda_dot(EMData &obj1, EMData &obj2) {
    float * h_v1 = obj1.get_data();
    float * h_v2 = obj2.get_data();
    int N = obj1.get_size();

    float * d_v1, * d_v2, * d_o;
    hipMallocManaged(&d_v1, N*sizeof(float));
    hipMallocManaged(&d_v2, N*sizeof(float));

    hipMemcpy(d_v1, h_v1, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, N*sizeof(float), hipMemcpyHostToDevice);

    int threads = THREADSPERBLOCK;
    int blocks = (N+threads-1)/threads;

    float *sum_ptr;
    hipError_t er = hipMallocManaged(&sum_ptr, blocks*sizeof(float));
    if (er != hipSuccess)
    {
        printf("1 %s\n",hipGetErrorString(er));
        exit(1);
    }

    kernel_dot<<<blocks,threads>>>(d_v1, d_v2, N, sum_ptr);
    hipDeviceSynchronize();
    hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("0 %s\n",hipGetErrorString(error));
        exit(1);
    }

    float sum = 0.0;
    for (int i = 0; i < blocks; ++i) {
        sum += sum_ptr[i];
    }

    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(sum_ptr);

    return sum;
}

float thrust_inner_product(EMData &obj1, EMData &obj2) {
    int N = obj1.get_size();
    float * d_ptr_1 = obj1.get_data();
    float * d_ptr_2 = obj2.get_data();
    DV d_v1(d_ptr_1, d_ptr_1+N);
    DV d_v2(d_ptr_2, d_ptr_2+N);
    
    return thrust::inner_product(d_v1.begin(),d_v1.end(),
                                 d_v2.begin(),
                                 0.0f);
}

float thrust_transform_reduce(EMData &obj1, EMData &obj2) {
    int  N = obj1.get_size();
    float * d_ptr_1 = obj1.get_data();
    float * d_ptr_2 = obj2.get_data();
    DV d_v1(d_ptr_1, d_ptr_1+N);
    DV d_v2(d_ptr_2, d_ptr_2+N);
    DV d_o(N);

    thrust::transform(d_v1.begin(),d_v1.end(),
                      d_v2.begin(),
                      d_o.begin(),
                      thrust::multiplies<float>());
    
    return thrust::reduce(d_o.begin(), d_o.end());
}

float cuda_cublas(EMData &obj1, EMData &obj2) {
    hipblasHandle_t handle;
    float * h_v1 = obj1.get_data();
    float * h_v2 = obj2.get_data();
    int N = obj1.get_size();

    float * d_v1, * d_v2, * d_o;
    hipMallocManaged(&d_v1, N*sizeof(float));
    hipMallocManaged(&d_v2, N*sizeof(float));

    hipMemcpy(d_v1, h_v1, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, N*sizeof(float), hipMemcpyHostToDevice);
    float *sum_ptr;
    hipError_t er = hipMallocManaged(&sum_ptr, sizeof(float));
    if (er != hipSuccess)
    {
        printf("1 %s\n",hipGetErrorString(er));
        exit(1);
    }

    *sum_ptr = 0.0f;


//    hipblasSdot (handle, N,
//                d_v1, 1,
//                d_v2, 1,
//                sum_ptr);

    hipblasDestroy(handle);

    float sum = *sum_ptr;
    hipFree(sum_ptr);

    return sum;
}
